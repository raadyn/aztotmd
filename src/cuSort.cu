#include "hip/hip_runtime.h"
#include "cuStruct.h"
#include "cuSort.h"
#include "utils.h"
#include "cuUtils.h"
//#include "cuTemp.h"

void alloc_sort(int nAt, int nCell, cudaMD* hmd)
{
    int flsize = sizeof(float) * nAt;
    int xyzsize = sizeof(float3) * nAt;
    int intsize = int_size * nAt;

    hipMalloc((void**)&hmd->sort_xyz, xyzsize);
    hipMalloc((void**)&hmd->sort_vls, xyzsize);
    hipMalloc((void**)&hmd->sort_frs, xyzsize);        //! íå âñåãäà, âîçìîæíî, ÷òî òîëüêî åñëè åñòü ñâÿçè (ò.å. åñòü ÷òî-òî, ÷òî îïðåäåëÿåò ñèëû ïåðåä ñîðòèðîâêîé)
    hipMalloc((void**)&hmd->sort_types, intsize);
    hipMalloc((void**)&hmd->sort_ind, intsize);
    hipMalloc((void**)&hmd->sort_parents, intsize);    //! íå âñåãäà
    hipMalloc((void**)&hmd->sort_nbonds, intsize);     //! íå âñåãäà
    hipMalloc((void**)&hmd->sort_nangles, intsize);    //! íå âñåãäà
    hipMalloc((void**)&hmd->sort_oldTypes, intsize);   //! íå âñåãäà
    hipMalloc((void**)&hmd->cellIndexes, intsize);     //? ìîæåò ýòî îòíîñèòñÿ ê cell list, à íå ê sort?
    hipMalloc((void**)&hmd->insideCellIndex, intsize); //? ìîæåò ýòî îòíîñèòñÿ ê cell list, à íå ê sort?
    hipMalloc((void**)&hmd->sort_masses, flsize);
    hipMalloc((void**)&hmd->sort_rMasshdT, flsize);
    hipMalloc((void**)(&hmd->sort_engs), flsize);          // for radiative thermostat only
    hipMalloc((void**)(&hmd->sort_radii), flsize);         // for radiative thermostat only
    hipMalloc((void**)(&hmd->sort_radstep), intsize);    // for radiative thermostat only

    // for trajectories output
    int* arr = (int*)malloc(intsize);
    hipMalloc((void**)(&hmd->sort_trajs), intsize);        
    int i;
    for (i = 0; i < nAt; i++)
        arr[i] = i;
    data_to_device((void**)(&hmd->sort_trajs), arr, intsize);

    //? ìîæåò ýòî îòíîñèòñÿ ê cell list, à íå ê sort?
    intsize = int_size * nCell;
    hipMalloc((void**)&hmd->firstAtomInCell, intsize);
    hipMalloc((void**)&hmd->nAtInCell, intsize);
}

void free_sort(cudaMD* hmd)
{
    hipFree(hmd->sort_xyz);
    hipFree(hmd->sort_vls);
    hipFree(hmd->sort_frs);        //! íå âñåãäà, âîçìîæíî, ÷òî òîëüêî åñëè åñòü ñâÿçè (ò.å. åñòü ÷òî-òî, ÷òî îïðåäåëÿåò ñèëû ïåðåä ñîðòèðîâêîé)
    hipFree(hmd->sort_types);
    hipFree(hmd->sort_ind);
    hipFree(hmd->sort_parents);    //! íå âñåãäà
    hipFree(hmd->sort_nbonds);     //! íå âñåãäà
    hipFree(hmd->sort_nangles);    //! íå âñåãäà    
    hipFree(hmd->sort_oldTypes);   //! íå âñåãäà
    hipFree(hmd->cellIndexes);     //? ìîæåò ýòî îòíîñèòñÿ ê cell list, à íå ê sort?
    hipFree(hmd->insideCellIndex); //? ìîæåò ýòî îòíîñèòñÿ ê cell list, à íå ê sort?
    hipFree(hmd->sort_masses);
    hipFree(hmd->sort_rMasshdT);
    hipFree(hmd->firstAtomInCell); //? ìîæåò ýòî îòíîñèòñÿ ê cell list, à íå ê sort?
    hipFree(hmd->nAtInCell);       //? ìîæåò ýòî îòíîñèòñÿ ê cell list, à íå ê sort?
    hipFree(hmd->sort_engs);    // for radiative thermostat only
    hipFree(hmd->sort_radii);    // for radiative thermostat only
    hipFree(hmd->sort_radstep);    // for radiative thermostat only
    hipFree(hmd->sort_trajs);        // for trajectories output
}

__device__ void switch_pointers(void** p1, void** p2)
{
    void* ptr = *p1;
    *p1 = *p2;
    *p2 = ptr;
}

__global__ void refresh_arrays(int use_bnd, int use_ang, cudaMD* md)
{
    //printf("beging refresh arrays\n");

    switch_pointers((void**)&(md->xyz), (void**)&(md->sort_xyz));
    switch_pointers((void**)&(md->vls), (void**)&(md->sort_vls));
    switch_pointers((void**)&(md->frs), (void**)&(md->sort_frs));   //! íå âñåãäà, âîçìîæíî, ÷òî òîëüêî åñëè åñòü ñâÿçè (ò.å. åñòü ÷òî-òî, ÷òî îïðåäåëÿåò ñèëû ïåðåä ñîðòèðîâêîé)
    switch_pointers((void**)&(md->types), (void**)&(md->sort_types));
    if (use_bnd)
    {
        //printf("switch_bnd\n");
        switch_pointers((void**)&(md->parents), (void**)&(md->sort_parents));
        switch_pointers((void**)&(md->nbonds), (void**)&(md->sort_nbonds));
        switch_pointers((void**)&(md->oldTypes), (void**)&(md->sort_oldTypes));
    }
    if (use_ang)
    {
        switch_pointers((void**)&(md->nangles), (void**)&(md->sort_nangles));
    }
    switch_pointers((void**)&(md->masses), (void**)&(md->sort_masses));
    switch_pointers((void**)&(md->rMasshdT), (void**)&(md->sort_rMasshdT));
    switch_pointers((void**)&(md->engs), (void**)&(md->sort_engs));    // for radiative thermostat only
    switch_pointers((void**)&(md->radii), (void**)&(md->radii));    // for radiative thermostat only
    switch_pointers((void**)&(md->radstep), (void**)&(md->sort_radstep));    // for radiative thermostat only

    //printf("end refresh arrays\n");
}

/* óæå åñòü â äðóãîì ìîäóëå
__global__ void clear_list(int cellPerBlock, int cellPerThread, cudaMD* md)
{
    int i;
    int id0 = blockIdx.x * cellPerBlock + threadIdx.x * cellPerThread;
    int N = min(id0 + cellPerThread, md->nCell);

    for (i = id0; i < N; i++)
        md->nAtInCell[i] = 0;
}
*/

__device__ void count_cell(int index, float3 xyz, cudaMD* md)
// save atom index with coordinates xyz in the cell list
{
    int c, j;

    c = floor((double)xyz.x * (double)md->cRevSize.x) * md->cnYZ + floor((double)xyz.y * (double)md->cRevSize.y) * md->cNumber.z + floor((double)xyz.z * (double)md->cRevSize.z);
    if (c >= md->nCell)
        printf("count cell: xyz=(%f; %f; %f)revsizes:[%f %f %f] c = %d\n", xyz.x, xyz.y, xyz.z, md->cRevSize.x, md->cRevSize.y, md->cRevSize.z, c);
    if (c < 0)
        printf("count cell: xyz=(%f; %f; %f) c = %d\n", xyz.x, xyz.y, xyz.z, c);

    md->cellIndexes[index] = c;
    j = atomicAdd(&(md->nAtInCell[c]), 1);    // increase the number of particles in cell[c] (it keeps in the 0th element of cell[cell_index] array)
    md->insideCellIndex[index] = j;
}

__global__ void calc_firstAtomInCell(cudaMD* md)
// define first index of atom in ordered array corresponding to each cell
//! ýòîò êîä ïî ñóòè ñåðèéíûé
{
    int i;
    int cnt = 0;
    //printf("start calc first atom\n");
    for (i = 0; i < md->nCell; i++)
    {
        md->firstAtomInCell[i] = cnt;
        cnt += md->nAtInCell[i];
    }
    //printf("end calc first atom\n");
}

__global__ void sort_atoms(int use_bnd, int use_ang, int atPerBlock, int atPerThread, cudaMD* md)
// sort atoms according to the cells belonging
{
    //printf("BEGIN SORT ATOMS(%d, %d)\n", blockIdx.x, threadIdx.x);
    int i, j;
    int id0 = blockIdx.x * atPerBlock + threadIdx.x * atPerThread;
    int N = min(id0 + atPerThread, md->nAt);
    for (i = id0; i < N; i++)
    {
        //if (i == 0)
          //  printf("begin sort atoms\n");

        // define new index:
        j = md->firstAtomInCell[md->cellIndexes[i]] + md->insideCellIndex[i];
        // copy data to ordered arrays:
        md->sort_xyz[j] = md->xyz[i];
        md->sort_vls[j] = md->vls[i];
        md->sort_frs[j] = md->frs[i];   //! íå âñåãäà, âîçìîæíî, ÷òî òîëüêî åñëè åñòü ñâÿçè (ò.å. åñòü ÷òî-òî, ÷òî îïðåäåëÿåò ñèëû ïåðåä ñîðòèðîâêîé)
        md->sort_types[j] = md->types[i];
        md->sort_masses[j] = md->masses[i];
        md->sort_rMasshdT[j] = md->rMasshdT[i];
        if (md->tstat == 2) // radiative thermostat /! òóò äîëæíà áûòü êîíñòàíòà ctTermRadi
        {
            md->sort_engs[j] = md->engs[i];    // for radiative thermostat only
            md->sort_radii[j] = md->radii[i];    // for radiative thermostat only
            md->sort_radstep[j] = md->radstep[i];    // for radiative thermostat only
        }
        //printf("SORT ATOMS bef use_bnd(%d, %d)\n", blockIdx.x, threadIdx.x);
        if (use_bnd)
        {
            // ñîðòèðîâêó ðîäèòèåëåé íóæíî äåëàòü â äâà äåéñòâèÿ, ñíà÷àëà ïåðåìåùàåì çíà÷åíèå ðîäèòåëÿ íà íîâîå ìåñòî
            // à ïîñêîëüêó çíà÷åíèå ðîäèòåëÿ âñå åù¸ â ñòàðîé òåðìèíîëîãèè, îòäåëüíûì êåðíåëîì ïåðåñ÷èòûâàåì åãî
            md->sort_parents[j] = md->parents[i];
            md->sort_nbonds[j] = md->nbonds[i];
        }
        //printf("SORT ATOMS bef use_ang(%d, %d)\n", blockIdx.x, threadIdx.x);
        if (use_ang)
        {
            md->sort_nangles[j] = md->nangles[i];
        }
        //printf("SORT ATOMS bef use_ang || use_bnd(%d, %d)\n", blockIdx.x, threadIdx.x);
        //! èëè åñëè èñïîëüçóåòñÿ âûâîä òðàåêòîðèé!
        if (use_bnd || use_ang)
        {
            md->sort_oldTypes[j] = md->oldTypes[i];
            md->sort_ind[i] = j;
        }
        //printf("SORT ATOMS aft use_ang || use_bnd(%d, %d)\n", blockIdx.x, threadIdx.x);

        //if (i == 0)
          //  printf("end sort atoms\n");
    }
}

__global__ void sort_parents_and_trajs(int atPerBlock, int atPerThread, cudaMD* md)
{
    int i;// , j;
    int id0 = blockIdx.x * atPerBlock + threadIdx.x * atPerThread;
    int N = min(id0 + atPerThread, md->nAt);
    for (i = id0; i < N; i++)
    {
        md->sort_parents[i] = md->sort_ind[md->sort_parents[i]];
        md->sort_trajs[i] = md->sort_ind[md->sort_trajs[i]];
    }
}


__global__ void sort_bonds(int bndPerBlock, int bndPerThread, cudaMD* md)
{
    int id0 = blockIdx.x * bndPerBlock + threadIdx.x * bndPerThread;
    int N = min(id0 + bndPerThread, md->nBond);
    int iBnd;
    for (iBnd = id0; iBnd < N; iBnd++)
    {
        md->bonds[iBnd].x = md->sort_ind[md->bonds[iBnd].x];
        md->bonds[iBnd].y = md->sort_ind[md->bonds[iBnd].y];
    }
}

__global__ void sort_angles(int angPerBlock, int angPerThread, cudaMD* md)
{
    //printf("start sort angles\n");
    int i;
    int id0 = blockIdx.x * angPerBlock + threadIdx.x * angPerThread;
    int N = min(id0 + angPerThread, md->nAngle);
    for (i = id0; i < N; i++)
    {
        md->angles[i].x = md->sort_ind[md->angles[i].x];
        md->angles[i].y = md->sort_ind[md->angles[i].y];
        md->angles[i].z = md->sort_ind[md->angles[i].z];
    }
}
